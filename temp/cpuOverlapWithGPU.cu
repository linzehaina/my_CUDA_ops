﻿#ifdef USE_DP
typedef double real;
#else
typedef float real;
#endif

#include <iostream>
#include <hip/hip_runtime.h>


//CUDA错误检测宏
#define CHECK(call)                                     \
    do                                                  \
    {                                                   \
        const hipError_t error_code = call;            \
        if (error_code != hipSuccess)                  \
        {                                               \
            printf("CUDA Error:\n");                    \
            printf("    File:       %s\n", __FILE__);   \
            printf("    Line:       %d\n", __LINE__);   \
            printf("    Error code: %d\n", error_code); \
            printf("    Error text: %s\n",              \
                   hipGetErrorString(error_code));     \
            exit(1);                                    \
        }                                               \
    } while (0)

const int NUM_REPEATS = 10;
const int N = 100000000;
const int M = sizeof(real) * N;
const int block_size = 128;
const int grid_size = (N - 1) / block_size + 1;

// 封装 timing 函数, 通过参数 ratio 与 overlap,
// 调节 cpu 计算的复杂度与是否重叠主机与设备的计算
void timing(const real *h_x,
            const real *h_y,
            real *h_z,
            const real *d_x,
            const real *d_y,
            real *d_z,
            const int ratio,
            bool overlap);

int main()
{
    real *h_x = (real *)malloc(M);
    real *h_y = (real *)malloc(M);
    real *h_z = (real *)malloc(M);
    for (int n = 0; n < N; ++n)
    {
        h_x[n] = 1.23;
        h_y[n] = 2.34;
    }

    real *d_x, *d_y, *d_z;
    CHECK(hipMalloc(&d_x, M));
    CHECK(hipMalloc(&d_y, M));
    CHECK(hipMalloc(&d_z, M));
    CHECK(hipMemcpy(d_x, h_x, M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_y, h_y, M, hipMemcpyHostToDevice));

    printf("Without CPU-GPU overlap (ratio = 10)\n");
    timing(h_x, h_y, h_z, d_x, d_y, d_z, 10, false);
    printf("With CPU-GPU overlap (ratio = 10)\n");
    timing(h_x, h_y, h_z, d_x, d_y, d_z, 10, true);

    printf("Without CPU-GPU overlap (ratio = 1)\n");
    timing(h_x, h_y, h_z, d_x, d_y, d_z, 1, false);
    printf("With CPU-GPU overlap (ratio = 1)\n");
    timing(h_x, h_y, h_z, d_x, d_y, d_z, 1, true);

    printf("Without CPU-GPU overlap (ratio = 1000)\n");
    timing(h_x, h_y, h_z, d_x, d_y, d_z, 1000, false);
    printf("With CPU-GPU overlap (ratio = 1000)\n");
    timing(h_x, h_y, h_z, d_x, d_y, d_z, 1000, true);

    free(h_x);
    free(h_y);
    free(h_z);
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(d_z));

    return 0;
}

void cpu_sum(const real *x, const real *y, real *z, const int N_host)
{
    for (int n = 0; n < N_host; ++n)
    {
        z[n] = x[n] + y[n];
    }
}

void __global__ gpu_sum(const real *x, const real *y, real *z)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        z[n] = x[n] + y[n];
    }
}

void timing(const real *h_x,
            const real *h_y,
            real *h_z,
            const real *d_x,
            const real *d_y,
            real *d_z,
            const int ratio,
            bool overlap)
{
    float t_sum = 0;
    float t2_sum = 0;

    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        hipEvent_t start, stop;
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        CHECK(hipEventRecord(start));
        hipEventQuery(start);

        // overlap == false, 意味不重叠计算, 所以先执行 cpu 计算, 设备会在此期间闲置等待
        // ratio 越高,  N / ratio 越小, cpu 执行的运算越少, 时间也就越短
        // ratio 可以用来调整 cpu 计算的时间长短, 以此对加速比进行实验
        if (!overlap)
        {
            cpu_sum(h_x, h_y, h_z, N / ratio);
        }

        gpu_sum<<<grid_size, block_size>>>(d_x, d_y, d_z);

        // overlap == true, 重叠主机计算与设备计算.
        if (overlap)
        {
            cpu_sum(h_x, h_y, h_z, N / ratio);
        }

        CHECK(hipEventRecord(stop));
        CHECK(hipEventSynchronize(stop));
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);

        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }

        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }

    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);
}
