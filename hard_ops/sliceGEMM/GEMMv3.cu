#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include "macro.h"
#include <iostream>
using namespace std;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])

float testError(
    void (*gpuSgemm)(float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K);
float testPerformance(
    void (*gpuSgemm)(float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat);

void cpuSgemm(
    float *a, float *b, float *c, const int M, const int N, const int K)
{

    for (int m = 0; m < M; m++)
    {
        for (int n = 0; n < N; n++)
        {
            float psum = 0.0;
            for (int k = 0; k < K; k++)
            {
                psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
            }
            c[OFFSET(m, n, N)] = psum;
        }
    }
}

//K must be a multiple of 16 
//N must be a multiple of 4
__global__ void sgemm_V3(
    float *__restrict__ a, float *__restrict__ b, float *__restrict__ c,
    const int M, const int N, const int K)
{

    const int BM = 128;
    const int BN = 128;
    const int BK = 8;
    const int TM = 8;
    const int TN = 8;

    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tid = ty * blockDim.x + tx;

    __shared__ float s_a[2][BK][BM];
    __shared__ float s_b[2][BK][BN];

    float r_load_a[4];
    float r_load_b[4];
    float r_comp_a[TM];
    float r_comp_b[TN];
    float r_c[TM][TN] = {0.0};

    int load_a_smem_m = tid >> 1;
    int load_a_smem_k = (tid & 1) << 2;
    int load_b_smem_k = tid >> 5;
    int load_b_smem_n = (tid & 31) << 2;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    // if(load_a_gmem_m >= M || load_b_gmem_n >= N || store_c_gmem_m >= M || store_c_gmem_n >= N)
    // {
    //     // printf("tid: %d\n", tid);
    //     return;
    // }

    //thread搬来的数据不一定是自己要用的数据

    // Load the first block of A and B matrices into shared memory
    {
        int load_a_gmem_k = load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);

        if(load_a_gmem_m < M)
            FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
        if(load_b_gmem_addr < N * K)
            FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

        s_a[0][load_a_smem_k][load_a_smem_m] = r_load_a[0];
        s_a[0][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[0][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[0][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[0][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);
    }

    //bug fixed: lack a sync
    __syncthreads();

    for (int bk = 1; bk < (K + BK - 1) / BK; bk++)
    {

        int smem_sel = (bk - 1) & 1;
        int smem_sel_next = bk & 1;

        int load_a_gmem_k = bk * BK + load_a_smem_k;
        int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_gmem_k, K);
        int load_b_gmem_k = bk * BK + load_b_smem_k;
        int load_b_gmem_addr = OFFSET(load_b_gmem_k, load_b_gmem_n, N);

        if(load_a_gmem_m < M)
            FLOAT4(r_load_a[0]) = FLOAT4(a[load_a_gmem_addr]);
        if(load_b_gmem_addr < N * K)
            FLOAT4(r_load_b[0]) = FLOAT4(b[load_b_gmem_addr]);

        #pragma unroll
        for (int tk = 0; tk < BK; tk++)
        {
            FLOAT4(r_comp_a[0]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2]);
            FLOAT4(r_comp_a[4]) = FLOAT4(s_a[smem_sel][tk][ty * TM / 2 + BM / 2]);
            FLOAT4(r_comp_b[0]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2]);
            FLOAT4(r_comp_b[4]) = FLOAT4(s_b[smem_sel][tk][tx * TN / 2 + BN / 2]);

            #pragma unroll
            for (int tm = 0; tm < TM; tm++)
            {
                #pragma unroll
                for (int tn = 0; tn < TN; tn++)
                {
                    r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
                }
            }
        }

        s_a[smem_sel_next][load_a_smem_k][load_a_smem_m] = r_load_a[0];
        s_a[smem_sel_next][load_a_smem_k + 1][load_a_smem_m] = r_load_a[1];
        s_a[smem_sel_next][load_a_smem_k + 2][load_a_smem_m] = r_load_a[2];
        s_a[smem_sel_next][load_a_smem_k + 3][load_a_smem_m] = r_load_a[3];
        FLOAT4(s_b[smem_sel_next][load_b_smem_k][load_b_smem_n]) = FLOAT4(r_load_b[0]);

        __syncthreads();
    }

    
    #pragma unroll
    for (int tk = 0; tk < BK; tk++)
    {
        FLOAT4(r_comp_a[0]) = FLOAT4(s_a[1][tk][ty * TM / 2]);
        FLOAT4(r_comp_a[4]) = FLOAT4(s_a[1][tk][ty * TM / 2 + BM / 2]);
        FLOAT4(r_comp_b[0]) = FLOAT4(s_b[1][tk][tx * TN / 2]);
        FLOAT4(r_comp_b[4]) = FLOAT4(s_b[1][tk][tx * TN / 2 + BN / 2]);

    #pragma unroll
        for (int tm = 0; tm < TM; tm++)
        {
            #pragma unroll
            for (int tn = 0; tn < TN; tn++)
            {
                r_c[tm][tn] += r_comp_a[tm] * r_comp_b[tn];
            }
        }
    }


    // bug fixed: the position of four parts of c need to be checked separately
    // Store results to global memory
    #pragma unroll
    for (int i = 0; i < TM / 2; i++)
    {
        int store_c_gmem_m = by * BM + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
        if(store_c_gmem_m < M && store_c_gmem_n < N)
        {
            FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i][0]);
            if(store_c_gmem_n + BN / 2 < N)
                FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i][4]);
        }
    }

    #pragma unroll
    for (int i = 0; i < TM / 2; i++)
    {
        int store_c_gmem_m = by * BM + BM / 2 + ty * TM / 2 + i;
        int store_c_gmem_n = bx * BN + tx * TN / 2;
        int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
        if(store_c_gmem_m < M && store_c_gmem_n < N)
        {
            FLOAT4(c[store_c_gmem_addr]) = FLOAT4(r_c[i + TM / 2][0]);
            if(store_c_gmem_n + BN / 2 < N)
                FLOAT4(c[store_c_gmem_addr + BN / 2]) = FLOAT4(r_c[i + TM / 2][4]);
        }
    }
}

int main(void)
{
    printf("\nKernal = sgemm_V3\n");
    const int outer_repeat = 10, inner_repeat = 1;
    const int BM = 128, BN = 128, TM = 8, TN = 8;
    void (*gpuSgemm)(float *, float *, float *, const int, const int, const int) = sgemm_V3;

    {
        const int M = 131, N = 132, K = 80;
        dim3 blockDim(BN / TN, BM / TM);
        dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
        float max_error = testError(gpuSgemm, gridDim, blockDim, M, N, K);
        printf("Max Error = %f\n", max_error);
    }

    const int M_list[14] = { 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 8000};
    const int N_list[14] = { 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 8000};
    const int K_list[14] = { 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 8000};
    
    const int TESTNUM = 13;
    for (int i = 0; i < TESTNUM; i++)
    {
        const int M = M_list[i], N = N_list[i], K = K_list[i];

        dim3 blockDim(BN / TN, BM / TM);
        dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int j = 0; j < outer_repeat; j++)
        {
            double this_sec = testPerformance(gpuSgemm, gridDim, blockDim, M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

        printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
    }
    return 0;
}

float testError(
    void (*gpuSgemm)(float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K)
{

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_d_c;
    h_a = (float *)malloc(size_a);
    h_b = (float *)malloc(size_b);
    h_c = (float *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (float *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = 1 * rand() / float(RAND_MAX);
    for (int i = 0; i < K * N; i++)
        h_b[i] = 1 * rand() / float(RAND_MAX);
    hipMemset(d_c, 15, size_c);

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gpuSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    CHECK(hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost));

    float max_error = 0.0;
    float max_error_percentage = 0.0;
    float avg1 = 0.0;
    float avg2 = 0.0;
    for (int i = 0; i < M * N; i++)
    {
        float this_error = abs(h_d_c[i] - h_c[i]);
        float this_error_percentage = abs(h_d_c[i] - h_c[i]) / h_c[i];
        avg1 += h_d_c[i];
        avg2 += h_c[i];
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);

        if(max_error_percentage != max_error_percentage || this_error_percentage != this_error_percentage)
            max_error_percentage = NAN;
        else {
            // if (this_error_percentage > max_error_percentage) {
            //     cout << "position: " << i / M << " " << i % N <<  " " << this_error_percentage << endl;
            //     cout << "h_d_c: " << h_d_c[i] << ", h_c: " << h_c[i] << endl;
            // }
            max_error_percentage = max(max_error_percentage, this_error_percentage);
        }
    }

    cout << "Average of GPU result: " << avg1 / (M*N)<< endl;
    cout << "Average of CPU result: " << avg2 / (M*N)<< endl;
    cout << "Max error percentage: " << max_error_percentage * 100 << "%" << endl;
    cout << "Max error: " << max_error << endl;

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_d_c);

    return max_error;
}

float testPerformance(
    void (*gpuSgemm)(float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat)
{

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    CHECK(hipEventRecord(start));
    for (int i = 0; i < repeat; i++)
        gpuSgemm<<<gridDim, blockDim>>> (d_a, d_b, d_c, M, N, K);
    CHECK(hipEventRecord(end));
    CHECK(hipEventSynchronize(end));

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return sec;
}