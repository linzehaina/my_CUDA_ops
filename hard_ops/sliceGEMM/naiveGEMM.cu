﻿#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;
// 将二维数组的行列索引转成一维数组的行列索引，这样可以更高效访问数据
// row, col：二维数组实际的行列索引，ld表示该数组实际的列数
// 例：二维数组实际的行列索引为(1, 3)，即第二行第四个元素，二维数据的总列数 = 5
// 返回的一位数组形式的索引为: 1*5 + 3 = 8
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

float testError(
    void (*gpuSgemm)(float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K);

// 定义naive gemm的kernel函数
__global__ void naiveSgemm(
    float * __restrict__ a, float * __restrict__ b, float * __restrict__ c,
    const int M, const int N, const int K) {
    
    // 当前thread在C矩阵中的row
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    // 当前thread在C矩阵中的col
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    while (m < M && n < N) {
        float psum = 0.0;
        // 告知编译器自动展开循环体，这样可以减少循环控制的开销（循环次数小的时候可以这么做）
        #pragma unroll
        // 取出A[row]和B[col]，然后逐个元素相乘累加，得到最终结果
        for (int k = 0; k < K; k++) {
            // a[OFFSET(m, k, K)]: 获取A[m][k]
            // b[OFFSET(k, n, N)]: 获取B[k][n]
            psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
        }
        c[OFFSET(m, n, N)] = psum;
        // 更新m和n，处理下一个元素
        n += blockDim.x * gridDim.x; // 每个block处理一行
        if (n >= N) {
            n = blockIdx.x * blockDim.x + threadIdx.x; // 重置n
            m += blockDim.y * gridDim.y; // 每个block处理一列
        }
    }
}

void cpuSgemm(
    float *a, float *b, float *c, const int M, const int N, const int K)
{

    for (int m = 0; m < M; m++)
    {
        for (int n = 0; n < N; n++)
        {
            float psum = 0.0;
            for (int k = 0; k < K; k++)
            {
                psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
            }
            c[OFFSET(m, n, N)] = psum;
        }
    }
}

int main() {
    const int BM = 32, BN = 32;
    const int m = 8000, n = 8000, k = 8000;

    {
        const int BM = 128, BN = 128, TM = 8, TN = 8;
        void (*gpuSgemm)(float *, float *, float *, const int, const int, const int) = naiveSgemm;
        const int M = 512, N = 512, K = 512;
        dim3 blockDim(BN, BM);
        dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
        float max_error = testError(gpuSgemm, gridDim, blockDim, M, N, K);
        printf("Max Error = %f\n", max_error);
    }

    // 定义线程块大小和网格大小
    dim3 blockDim(BN, BM);
    dim3 gridDim((n + BN - 1) / BN, (m + BM - 1) / BM);

    // 计算矩阵大小（以字节为单位）
    size_t sizeA = m * k * sizeof(float);
    size_t sizeB = k * n * sizeof(float);
    size_t sizeC = m * n * sizeof(float);

    // 分配主机内存
    float* h_A = (float*)malloc(sizeA);
    float* h_B = (float*)malloc(sizeB);
    float* h_C = (float*)malloc(sizeC);

    // 初始化矩阵 A 和 B
    srand(static_cast<unsigned int>(time(0)));
    for (int i = 0; i < m * k; i++) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
    }
    for (int i = 0; i < k * n; i++) {
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // 分配设备内存
    float* d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    // 创建 CUDA 事件以测量执行时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 启动计时
    hipEventRecord(start);

    // 调用 CUDA 矩阵乘法内核
    naiveSgemm<<<gridDim, blockDim>>>(d_A, d_B, d_C, m, n, k);

    // 停止计时
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // 将结果从设备复制回主机
    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    // 计算并打印执行时间
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUDA Matrix Multiplication Time: " << milliseconds << " ms" << std::endl;

    // 释放资源
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}


float testError(
    void (*gpuSgemm)(float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K)
{

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_d_c;
    h_a = (float *)malloc(size_a);
    h_b = (float *)malloc(size_b);
    h_c = (float *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (float *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0; i < K * N; i++)
        h_b[i] = rand() / float(RAND_MAX);
    hipMemset(d_c, 15, size_c);

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gpuSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    float avg1 = 0.0;
    float avg2 = 0.0;
    for (int i = 0; i < M * N; i++)
    {
        float this_error = abs(h_d_c[i] - h_c[i]);
        avg1 += h_d_c[i];
        avg2 += h_c[i];
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    cout << "Average of GPU result: " << avg1 / (M*N)<< endl;
    cout << "Average of CPU result: " << avg2 / (M*N)<< endl;

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_d_c);

    return max_error;
}